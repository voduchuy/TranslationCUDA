#include "hip/hip_runtime.h"
//
// Created by huy on 12/14/19.
//
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include <hiprand/hiprand_kernel.h>

#define CUDACHKERR() { \
hipError_t ierr = hipGetLastError();\
if (ierr != hipSuccess){ \
    printf("%s in %s at line %d\n", hipGetErrorString(ierr), __FILE__, __LINE__);\
    exit(EXIT_FAILURE); \
}\
}\


__global__
void init_rand_states(hiprandState_t *rstates) {
  hiprand_init(0, blockIdx.x, 0, &rstates[blockIdx.x]);
}

__global__
void initialize_ribosome_locations(const int num_rib, int *X) {
  const uint &thread_id = threadIdx.x;
  const uint &sample_id = blockIdx.x;

  uint ncodon_loc = num_rib / blockDim.x;
  uint idx_start = sample_id * num_rib + thread_id * ncodon_loc;
  for (int i{0}; i < ncodon_loc; ++i) {
    X[idx_start + i] = 0;
  }

  if (thread_id < num_rib % blockDim.x) {
    X[sample_id * num_rib + blockDim.x * ncodon_loc + thread_id] = 0;
  }

}

__device__
int draw_uniforms(hiprandState_t *rstate, double *rn) {
  rn[0] = hiprand_uniform_double(rstate);
  rn[1] = hiprand_uniform_double(rstate);
}

__device__
void shift_arrays(const int to_shift, const uint n, int *x_shared, int *x_shared_copy) {
  // copy x[to_shift:] to x_copy
  uint n_passes = (n - to_shift) / blockDim.x;
  for (uint k{0}; k < n_passes; ++k) {
    x_shared_copy[k * blockDim.x + threadIdx.x] = x_shared[to_shift + k * blockDim.x + threadIdx.x];
  }
  if (threadIdx.x < (n - to_shift) % blockDim.x) {
    x_shared_copy[n_passes * blockDim.x + threadIdx.x] = x_shared[to_shift + n_passes * blockDim.x + threadIdx.x];
  }
  // make everything in x zero
  n_passes = n / blockDim.x;
  for (uint k{0}; k < n_passes; ++k) {
    x_shared[k * blockDim.x + threadIdx.x] = 0;
  }
  if (threadIdx.x < n % blockDim.x) {
    x_shared[n_passes * blockDim.x + threadIdx.x] = 0;
  }
  // copy back from x_copy to x
  n_passes = (n - to_shift) / blockDim.x;
  for (uint k{0}; k < n_passes; ++k) {
    x_shared[k * blockDim.x + threadIdx.x] = x_shared_copy[k * blockDim.x + threadIdx.x];
  }
  if (threadIdx.x < (n - to_shift) % blockDim.x) {
    x_shared[n_passes * blockDim.x + threadIdx.x] = x_shared_copy[n_passes * blockDim.x + threadIdx.x];
  }
}

__global__
void update_state(const int num_times,
                  const double *t_array,
                  int num_excl,
                  int gene_len,
                  int num_rib,
                  int *X,
                  hiprandState_t *rstates,
                  const double *rates,
                  const int *probe_design,
                  int *intensity) {
  const uint &thread_id = threadIdx.x;
  const uint &sample_id = blockIdx.x;

  extern __shared__ double wsp[];
  // Partition the shared memory into appropriate arrays
  double *rn = wsp;
  double *t_now_ptr = wsp + 2;
  double *stepsize_ptr = t_now_ptr + 1;
  double *propensities = stepsize_ptr + 1;
  int *x_shared = ( int * ) (propensities + num_rib);
  int *x_wsp = x_shared + num_rib;
  int *to_shift_ptr = x_wsp + num_rib;
  int *idx_to_output = to_shift_ptr + 1;

  int *current_intensity;
  double &t_now = *t_now_ptr;
  double &stepsize = *stepsize_ptr;
  int &to_shift = *to_shift_ptr;
  const double &t_final = t_array[num_times - 1];

  // INITIALIZARION
  // Init time
  if (thread_id == 0) {
    t_now = 0.0;
    to_shift = 0;
    *idx_to_output = 0;
  }

  // Copy initial ribosome locations to shared memory
  uint idx;
  uint n_passes = num_rib / blockDim.x;

  for (uint k{0}; k < n_passes; ++k) {
    x_shared[k * blockDim.x + thread_id] = X[sample_id * num_rib + k * blockDim.x + thread_id];
  }
  if (thread_id < num_rib % blockDim.x) {
    x_shared[n_passes * blockDim.x + thread_id] = X[sample_id * num_rib + n_passes * blockDim.x + thread_id];
  }


  // STEPPING
  while (t_now < t_final) {
    // Compute current intensity
    n_passes = num_rib / blockDim.x;
    for (uint k{0}; k < n_passes; ++k) {
      idx = k * blockDim.x + thread_id;
      x_wsp[idx] = probe_design[x_shared[idx]];
    }
    if (thread_id < num_rib % blockDim.x) {
      idx = num_rib - num_rib % blockDim.x + thread_id;
      x_wsp[idx] = probe_design[x_shared[idx]];
    }
    thrust::inclusive_scan(thrust::device, x_wsp, x_wsp + num_rib, x_wsp);
    current_intensity = x_wsp + num_rib - 1;
    // copy current intensity to appropriate locations in global memory
    if (thread_id == 0) {
      while (t_array[*idx_to_output] <= t_now) {
        intensity[sample_id * num_times + (*idx_to_output)] = (*current_intensity);
        (*idx_to_output)++;
        if (*idx_to_output >= num_times) {
          break;
        }
      }
    }
    // compute propensities
    n_passes = num_rib / blockDim.x;
    for (uint k{0}; k < n_passes; ++k) {
      idx = k * blockDim.x + thread_id;
      propensities[idx] = rates[x_shared[idx]] * (
          (idx == 0)
              +
                  (idx != 0)
                      *
                          (x_shared[idx - 1] - x_shared[idx] > num_excl)
      );
    }
    if (thread_id < num_rib % blockDim.x) {
      idx = num_rib - num_rib % blockDim.x + thread_id;
      propensities[idx] = rates[x_shared[idx]] *
          (
              (idx == 0)
                  +
                      (idx != 0)
                          *
                              (x_shared[idx - 1] - x_shared[idx] > num_excl)
          );

    }

    // transform the propensities array to its cumsum array
    thrust::inclusive_scan(thrust::device, propensities, propensities + num_rib, propensities);

    // determine stepsize and the next ribosome to move
    if (thread_id == 0) {
      draw_uniforms(rstates + sample_id, rn);
      stepsize = -1.0 * log(rn[0]) / propensities[num_rib - 1];
    }
    t_now += stepsize;
    // update state
    if (thread_id == 0) {
      if (t_now + stepsize <= t_final) {
        for (int i{0}; i < num_rib; ++i) {
          if (propensities[i] >= rn[1] * propensities[num_rib - 1]) {
            x_shared[i] = (x_shared[i] + 1) % (gene_len + 1);
            if (x_shared[i] == 0) {
              to_shift = i + 1;
            }
            break;
          }
        }
      }
    }
    // check if we need to shift ribosomes locations so that the first nonzero is at the beginning
    if (to_shift > 0) {
      shift_arrays(to_shift, num_rib, x_shared, x_wsp);
      if (thread_id == 0) {
        to_shift = 0;
      }
    }
  }
  // Compute intensity at final time
  n_passes = num_rib / blockDim.x;
  for (uint k{0}; k < n_passes; ++k) {
    idx = k * blockDim.x + thread_id;
    x_wsp[idx] = probe_design[x_shared[idx]];
  }
  if (thread_id < num_rib % blockDim.x) {
    idx = num_rib - num_rib % blockDim.x + thread_id;
    x_wsp[idx] = probe_design[x_shared[idx]];
  }
  thrust::inclusive_scan(thrust::device, x_wsp, x_wsp + num_rib, x_wsp);
  current_intensity = x_wsp + num_rib - 1;
  // copy current intensity to appropriate locations in global memory
  if (thread_id == 0) {
    intensity[sample_id * num_times + num_times - 1] = (*current_intensity);
  }

// COPY FINAL RIBOSOMES LOCATIONS TO GLOBAL MEMORY
  n_passes = num_rib / blockDim.x;
  for (
      int k{0};
      k < n_passes;
      ++k) {
    idx = thread_id + k * blockDim.x;
    X[
        sample_id * num_rib
            + idx] = x_shared[idx];
  }
  if (thread_id < num_rib % blockDim.x) {
    X[
        sample_id * num_rib
            + blockDim.
                x * n_passes
            + thread_id] = x_shared[blockDim.
        x * n_passes
        + thread_id];
  }
}

int main(int argc, char **argv) {
  const int num_rib_max = 64;
  const int num_samples = 10;
  const int n_excl = 3;
  const int gene_len = 10;

  const int num_times = 5;
  const double t_array[] = {0.0, 10.0, 20.0, 50.0, 1000.0};

  thrust::device_vector<double> rates(gene_len + 1, 1.0);
  thrust::device_vector<int> probe_design(gene_len + 1);
  thrust::device_vector<int> intensity(num_times * num_samples);

  thrust::fill(probe_design.begin() + 1, probe_design.end(), 1);
  thrust::fill(intensity.begin(), intensity.end(), 0);

  thrust::device_vector<double> t_array_device(5);
  thrust::copy(&t_array[0], &t_array[5], t_array_device.begin());

  hiprandState_t *rand_states;
  hipMalloc(( void ** ) &rand_states, num_samples * sizeof(hiprandState_t));
  CUDACHKERR();

  init_rand_states<<<num_samples, 1>>>(rand_states);
  CUDACHKERR();

  int *X;
  hipMalloc(( void ** ) &X, num_samples * num_rib_max * sizeof(int));
  CUDACHKERR();
  initialize_ribosome_locations<<<num_samples, 32, 0>>>(num_rib_max, X);
  CUDACHKERR();
  size_t shared_mem_size = 2 * sizeof(double) // for the two uniform random numbers
      + 2 * sizeof(double) // for time and stepsize
      + num_rib_max * sizeof(double) // for propensities
      + num_rib_max * sizeof(int) // for ribosome locations
      + num_rib_max * sizeof(int) // temporary space to copy ribosome locations (when shifting)
      + sizeof(int) // amount to shift
      + sizeof(int) // current intensity
      + sizeof(int) // time array index to output the intesnity to
  ;
  update_state<<<num_samples, 32, shared_mem_size>>>(num_times,
                                                     thrust::raw_pointer_cast(&t_array_device[0]),
                                                     n_excl,
                                                     gene_len,
                                                     num_rib_max,
                                                     X,
                                                     rand_states,
                                                     thrust::raw_pointer_cast(&rates[0]),
                                                     thrust::raw_pointer_cast(&probe_design[0]),
                                                     thrust::raw_pointer_cast(&intensity[0]));
  CUDACHKERR();

  int X_host[num_samples][num_rib_max];
  hipMemcpy(( void * ) X_host, ( void * ) X, num_samples * num_rib_max * sizeof(int), hipMemcpyDeviceToHost);
  CUDACHKERR();

  thrust::host_vector<int> intensity_host(num_samples * num_times);
  thrust::copy(intensity.begin(), intensity.end(), intensity_host.begin());

  for (int i{0}; i < num_samples; ++i) {
    for (int j{0}; j < num_rib_max; ++j) {
      std::cout << X_host[i][j] << " ";
    }
    std::cout << "\n";
    for (int j{0}; j < num_times; ++j) {
      std::cout << intensity_host[i * num_times + j] << " ";
    }
    std::cout << "\n";
  }

  hipFree(X);
  CUDACHKERR();
  hipFree(rand_states);
  CUDACHKERR();
  return 0;
}