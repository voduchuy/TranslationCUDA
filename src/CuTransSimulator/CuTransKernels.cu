#include "hip/hip_runtime.h"
//
// Created by huy on 5/10/20.
//

#include "CuTransKernels.h"

namespace ssit {

__global__
void init_rand_states(hiprandState_t *rstates, int seed = 0) {
  hiprand_init(seed, blockIdx.x*blockDim.x + threadIdx.x, 0, &rstates[blockIdx.x*blockDim.x + threadIdx.x]);
}

__global__
void update_state(const int num_times,
                  const double *t_array,
                  int num_excl,
                  int gene_len,
                  int num_rib_max,
                  int *X,
                  hiprandState_t *rstates,
                  const double *rates,
                  const int *probe_design,
                  int *intensity) {
  const uint &thread_id = threadIdx.x;
  const uint &sample_id = blockIdx.x;
  hiprandState_t rstate_loc = rstates[sample_id*blockDim.x + thread_id];

  extern __shared__ double wsp[];
  // Partition the shared memory into appropriate arrays
  double *rn = wsp;
  double *t_now_ptr = wsp + blockDim.x;
  double *stepsize_ptr = t_now_ptr + 1;
  double *doub_wsp = stepsize_ptr + 1;
  double *rate_caches = doub_wsp + num_rib_max;
  int *x_shared = ( int * ) (rate_caches + num_rib_max);
  int *x_wsp = x_shared + num_rib_max;
  int *c_caches = x_wsp + num_rib_max;
  int *to_shift_ptr = c_caches + num_rib_max;
  int *idx_to_output = to_shift_ptr + 1;
  int *n_active_ptr = idx_to_output + 1;
  int *i_random_ptr = n_active_ptr + 1;

  int *current_intensity;
  double &t_now = *t_now_ptr;
  double &stepsize = *stepsize_ptr;
  int &to_shift = *to_shift_ptr;
  const double &t_final = t_array[num_times - 1];
  int &n_active = *n_active_ptr;
  int &i_random = *i_random_ptr;

  // INITIALIZATION
  // Init time
  if (thread_id == 0) {
    t_now = 0.0;
    to_shift = 0;
    *idx_to_output = 0;
  }

  // Copy initial ribosome locations and rates to shared memory
  uint idx;
  uint k{0};
  while ((idx = k * blockDim.x + thread_id) < num_rib_max) {
    x_shared[idx] = X[sample_id * num_rib_max + idx];
    rate_caches[idx] = rates[x_shared[idx]];
    c_caches[idx] = probe_design[x_shared[idx]];
    k++;
  }
  __syncthreads();
  if (thread_id == 0) {
    n_active = 0;
    for (int i{0}; i < num_rib_max; ++i) {
      if (x_shared[i] > 0) n_active++;
    }
    n_active++;
  }
  __syncthreads();
  // Draw the first cache of random numbers
  rn[thread_id] = hiprand_uniform_double(&rstate_loc);
  if (thread_id == 0) i_random = 0;
  __syncthreads();
  // STEPPING
  while (t_now < t_final) {
    // Compute current intensity
    k = 0;
    while ((idx = k * blockDim.x + thread_id) < n_active) {
      x_wsp[idx] = c_caches[idx];
      k++;
    }
    __syncthreads();
    if (thread_id == 0) thrust::inclusive_scan(thrust::seq, x_wsp, x_wsp + n_active, x_wsp);
    __syncthreads();
    if (thread_id == 0) current_intensity = x_wsp + n_active - 1;
    // copy current intensity to appropriate locations in global memory
    if (thread_id == 0) {
      while (t_array[*idx_to_output] <= t_now) {
        intensity[sample_id * num_times + (*idx_to_output)] = (*current_intensity);
        (*idx_to_output)++;
        if (*idx_to_output >= num_times) {
          break;
        }
      }
    }
    __syncthreads();

    // compute propensities
    k = 0;
    while ((idx = k * blockDim.x + thread_id) < n_active) {
      doub_wsp[idx] = rate_caches[idx] * (
          (idx == 0) + (idx != 0) * (x_shared[idx - 1] - x_shared[idx] > num_excl)
      );
      k++;
    }
    __syncthreads();

    // transform the propensities array to its cumsum array
    if (thread_id == 0) {
      thrust::inclusive_scan(thrust::seq, doub_wsp, doub_wsp + n_active, doub_wsp);
    }
    __syncthreads();

    // determine stepsize
    if (thread_id == 0) {
      stepsize = log(1.0/rn[i_random]) / doub_wsp[n_active - 1];
    }

    // choose the ribosome to move
    k = 0;
    while ((idx = k * blockDim.x + thread_id) < n_active) {
      x_wsp[idx] = (doub_wsp[idx] >= rn[i_random + 1] * doub_wsp[n_active - 1]);
      k++;
    }
    __syncthreads();

    if (thread_id == 0) idx = thrust::find(thrust::seq, x_wsp, x_wsp + n_active, 1) - x_wsp;
    __syncthreads();

    if (thread_id == 0) {
      if (t_now + stepsize <= t_final) {
        t_now += stepsize;

        x_shared[idx] = (x_shared[idx] + 1) % (gene_len + 1);
        if ((idx == n_active - 1) & (n_active < num_rib_max)) {
          n_active++;
          rate_caches[n_active - 1] = rates[0];
          c_caches[n_active - 1] = probe_design[0];
        } else if (x_shared[idx] == 0) {
          to_shift = ( int ) 1;
        }

        // update rates and probe design coefficients
        rate_caches[idx] = rates[x_shared[idx]];
        c_caches[idx] = probe_design[x_shared[idx]];
      } else {
        t_now = t_final;
      }
    }
    __syncthreads();

    // check if we need to shift ribosomes locations so that the first nonzero is at the beginning
    if (to_shift > 0) {
      _blockwise_shift_arrays(1, n_active, x_shared, x_wsp);
      _blockwise_shift_arrays(1, n_active, c_caches, x_wsp);
      _blockwise_shift_arrays(1, n_active, rate_caches, doub_wsp);
      n_active = max(1, n_active-1);
      if (thread_id == 0) {
        to_shift = 0;
      }
    }
    __syncthreads();
    // Pop out the used random numbers and check that we have enough random numbers in store
    if (thread_id == 0) i_random += 2;
    __syncthreads();
    if (i_random >= blockDim.x){
      rn[thread_id] = hiprand_uniform_double(&rstate_loc);
      __syncthreads();
      if (thread_id == 0) i_random = 0;
    }
    __syncthreads();
  }

  // Compute current intensity
  k = 0;
  while ((idx = k * blockDim.x + thread_id) < n_active) {
    x_wsp[idx] = c_caches[idx];
    k++;
  }
  __syncthreads();
  if (thread_id == 0) thrust::inclusive_scan(thrust::seq, x_wsp, x_wsp + n_active, x_wsp);
  __syncthreads();
  if (thread_id == 0) current_intensity = x_wsp + n_active - 1;
  // copy current intensity to appropriate locations in global memory
  if (thread_id == 0 && (*idx_to_output < num_times)) {
    while (t_array[*idx_to_output] <= t_now) {
      intensity[sample_id * num_times + (*idx_to_output)] = (*current_intensity);
      (*idx_to_output)++;
      if (*idx_to_output >= num_times) {
        break;
      }
    }
  }
  __syncthreads();

// COPY FINAL RIBOSOMES LOCATIONS TO GLOBAL MEMORY
  k = 0;
  while ((idx = k * blockDim.x + thread_id) < num_rib_max) {
    X[sample_id * num_rib_max + idx] = x_shared[idx];
    k++;
  }
// Update random states
  rstates[sample_id*blockDim.x + thread_id] = rstate_loc;
  __syncthreads();
}
}